#include "hip/hip_runtime.h"
/*
 * yolo_layer.cu
 *
 * This code was originally written by wang-xinyu under MIT license.
 * I took it from:
 *
 *     https://github.com/wang-xinyu/tensorrtx/tree/master/yolov4
 *
 * and made necessary modifications.
 *
 * - JK Jung
 */

#include "yolo_layer.h"

using namespace Yolo;

namespace
{
// Write values into buffer
template <typename T>
void write(char*& buffer, const T& val)
{
    *reinterpret_cast<T*>(buffer) = val;
    buffer += sizeof(T);
}

// Read values from buffer
template <typename T>
void read(const char*& buffer, T& val)
{
    val = *reinterpret_cast<const T*>(buffer);
    buffer += sizeof(T);
}
} // namespace

namespace nvinfer1
{
    YoloPluginDynamic::YoloPluginDynamic(const std::string name, int yolo_width, int yolo_height, int num_anchors, float* anchors, int num_classes, int input_width, int input_height, float scale_x_y, int new_coords)
        : mLayerName(name)
    {
        mYoloWidth   = yolo_width;
        mYoloHeight  = yolo_height;
        mNumAnchors  = num_anchors;
        memcpy(mAnchorsHost, anchors, num_anchors * 2 * sizeof(float));
        mNumClasses  = num_classes;
        mInputWidth  = input_width;
        mInputHeight = input_height;
        mScaleXY     = scale_x_y;
        mNewCoords   = new_coords;

        CHECK(hipMalloc(&mAnchors, MAX_ANCHORS * 2 * sizeof(float)));
        CHECK(hipMemcpy(mAnchors, mAnchorsHost, mNumAnchors * 2 * sizeof(float), hipMemcpyHostToDevice));
    }

    YoloPluginDynamic::YoloPluginDynamic(const std::string name, const void* data, size_t length)
        : mLayerName(name)
    {
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mThreadCount);
        read(d, mYoloWidth);
        read(d, mYoloHeight);
        read(d, mNumAnchors);
        memcpy(mAnchorsHost, d, MAX_ANCHORS * 2 * sizeof(float));
        d += MAX_ANCHORS * 2 * sizeof(float);
        read(d, mNumClasses);
        read(d, mInputWidth);
        read(d, mInputHeight);
        read(d, mScaleXY);
        read(d, mNewCoords);

        CHECK(hipMalloc(&mAnchors, MAX_ANCHORS * 2 * sizeof(float)));
        CHECK(hipMemcpy(mAnchors, mAnchorsHost, mNumAnchors * 2 * sizeof(float), hipMemcpyHostToDevice));

        ASSERT(d == a + length);
    }

    void YoloPluginDynamic::serialize(void* buffer) const
    {
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mThreadCount);
        write(d, mYoloWidth);
        write(d, mYoloHeight);
        write(d, mNumAnchors);
        memcpy(d, mAnchorsHost, MAX_ANCHORS * 2 * sizeof(float));
        d += MAX_ANCHORS * 2 * sizeof(float);
        write(d, mNumClasses);
        write(d, mInputWidth);
        write(d, mInputHeight);
        write(d, mScaleXY);
        write(d, mNewCoords);

        ASSERT(d == a + getSerializationSize());
    }

    size_t YoloPluginDynamic::getSerializationSize() const
    {
        return sizeof(mThreadCount) + \
               sizeof(mYoloWidth) + sizeof(mYoloHeight) + \
               sizeof(mNumAnchors) + MAX_ANCHORS * 2 * sizeof(float) + \
               sizeof(mNumClasses) + \
               sizeof(mInputWidth) + sizeof(mInputHeight) + \
               sizeof(mScaleXY) + sizeof(mNewCoords);
    }

    IPluginV2DynamicExt* YoloPluginDynamic::clone() const
    {
        YoloPluginDynamic *p = new YoloPluginDynamic(mLayerName, mYoloWidth, mYoloHeight, mNumAnchors, (float*) mAnchorsHost, mNumClasses, mInputWidth, mInputHeight, mScaleXY, mNewCoords);
        p->setPluginNamespace(mPluginNamespace.c_str());
        return p;
    }

    DimsExprs YoloPluginDynamic::getOutputDimensions(int outputIndex, const DimsExprs* inputs, int nbInputs, IExprBuilder& exprBuilder)
    {
        ASSERT(nbInputs == 1);
        ASSERT(outputIndex == 0);
        ASSERT(inputs[0].d[1] == exprBuilder.constant((mNumClasses + 5) * mNumAnchors));
        ASSERT(inputs[0].d[2] == exprBuilder.constant(mYoloHeight));
        ASSERT(inputs[0].d[3] == exprBuilder.constant(mYoloWidth));

        // output detection results to the channel dimension
        int totalsize = mYoloWidth * mYoloHeight * mNumAnchors * sizeof(Detection) / sizeof(float);

        DimsExprs ret;
        ret.nbDims = 4;
        ret.d[0] = inputs[0].d[0];  // batch_size
        ret.d[1] = exprBuilder.constant(totalsize);
        ret.d[2] = exprBuilder.constant(1);
        ret.d[3] = exprBuilder.constant(1);
        return ret;
    }

    bool YoloPluginDynamic::supportsFormatCombination(int pos, const PluginTensorDesc* inOut, int nbInputs, int nbOutputs)
    {
        ASSERT(nbInputs == 1);
        ASSERT(nbOutputs == 1);
        return inOut[pos].format == TensorFormat::kLINEAR && inOut[pos].type == DataType::kFLOAT;
    }

    // Return the DataType of the plugin output at the requested index
    DataType YoloPluginDynamic::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
        ASSERT(index == 0);
        ASSERT(nbInputs == 1);
        ASSERT(inputTypes[0] == DataType::kFLOAT);
        return DataType::kFLOAT;
    }

    const char* YoloPluginDynamic::getPluginType() const
    {
        return "YoloPluginDynamic_TRT";
    }

    const char* YoloPluginDynamic::getPluginVersion() const
    {
        return "1";
    }

    int YoloPluginDynamic::initialize()
    {
        return 0;
    }

    void YoloPluginDynamic::terminate()
    {
        CHECK(hipFree(mAnchors));
    }

    void YoloPluginDynamic::destroy()
    {
        delete this;
    }

    void YoloPluginDynamic::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* YoloPluginDynamic::getPluginNamespace() const
    {
        return mPluginNamespace.c_str();
    }

    inline __device__ float sigmoidGPU(float x) { return 1.0f / (1.0f + __expf(-x)); }

    inline __device__ float scale_sigmoidGPU(float x, float s)
    {
        return s * sigmoidGPU(x) - (s - 1.0f) * 0.5f;
    }

    // CalDetection(): This kernel processes 1 yolo layer calculation.  It
    // distributes calculations so that 1 GPU thread would be responsible
    // for each grid/anchor combination.
    // NOTE: The output (x, y, w, h) are between 0.0 and 1.0
    //       (relative to orginal image width and height).
    __global__ void CalDetection(const float *input, float *output,
                                 int batch_size,
                                 int yolo_width, int yolo_height,
                                 int num_anchors, const float *anchors,
                                 int num_classes, int input_w, int input_h,
                                 float scale_x_y)
    {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        Detection* det = ((Detection*) output) + idx;
        int total_grids = yolo_width * yolo_height;
        if (idx >= batch_size * total_grids * num_anchors) return;

        int info_len = 5 + num_classes;
        //int batch_idx = idx / (total_grids * num_anchors);
        int group_idx = idx / total_grids;
        int anchor_idx = group_idx % num_anchors;
        const float* cur_input = input + group_idx * (info_len * total_grids) + (idx % total_grids);

        int class_id;
        float max_cls_logit = -HIP_INF_F;  // minus infinity
        for (int i = 5; i < info_len; ++i) {
            float l = *(cur_input + i * total_grids);
            if (l > max_cls_logit) {
                max_cls_logit = l;
                class_id = i - 5;
            }
        }
        float max_cls_prob = sigmoidGPU(max_cls_logit);
        float box_prob = sigmoidGPU(*(cur_input + 4 * total_grids));
        //if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH)
        //    return;

        int row = (idx % total_grids) / yolo_width;
        int col = (idx % total_grids) % yolo_width;

        det->bbox[0] = (col + scale_sigmoidGPU(*(cur_input + 0 * total_grids), scale_x_y)) / yolo_width;    // [0, 1]
        det->bbox[1] = (row + scale_sigmoidGPU(*(cur_input + 1 * total_grids), scale_x_y)) / yolo_height;   // [0, 1]
        det->bbox[2] = __expf(*(cur_input + 2 * total_grids)) * *(anchors + 2 * anchor_idx + 0) / input_w;  // [0, 1]
        det->bbox[3] = __expf(*(cur_input + 3 * total_grids)) * *(anchors + 2 * anchor_idx + 1) / input_h;  // [0, 1]

        det->bbox[0] -= det->bbox[2] / 2;  // shift from center to top-left
        det->bbox[1] -= det->bbox[3] / 2;

        det->det_confidence = box_prob;
        det->class_id = class_id;
        det->class_confidence = max_cls_prob;
    }

    inline __device__ float scale(float x, float s)
    {
        return s * x - (s - 1.0f) * 0.5f;
    }

    inline __device__ float square(float x)
    {
        return x * x;
    }

    __global__ void CalDetection_NewCoords(const float *input, float *output,
                                           int batch_size,
                                           int yolo_width, int yolo_height,
                                           int num_anchors, const float *anchors,
                                           int num_classes, int input_w, int input_h,
                                           float scale_x_y)
    {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        Detection* det = ((Detection*) output) + idx;
        int total_grids = yolo_width * yolo_height;
        if (idx >= batch_size * total_grids * num_anchors) return;

        int info_len = 5 + num_classes;
        //int batch_idx = idx / (total_grids * num_anchors);
        int group_idx = idx / total_grids;
        int anchor_idx = group_idx % num_anchors;
        const float* cur_input = input + group_idx * (info_len * total_grids) + (idx % total_grids);

        int class_id;
        float max_cls_prob = -HIP_INF_F;  // minus infinity
        for (int i = 5; i < info_len; ++i) {
            float l = *(cur_input + i * total_grids);
            if (l > max_cls_prob) {
                max_cls_prob = l;
                class_id = i - 5;
            }
        }
        float box_prob = *(cur_input + 4 * total_grids);
        //if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH)
        //    return;

        int row = (idx % total_grids) / yolo_width;
        int col = (idx % total_grids) % yolo_width;

        det->bbox[0] = (col + scale(*(cur_input + 0 * total_grids), scale_x_y)) / yolo_width;                   // [0, 1]
        det->bbox[1] = (row + scale(*(cur_input + 1 * total_grids), scale_x_y)) / yolo_height;                  // [0, 1]
        det->bbox[2] = square(*(cur_input + 2 * total_grids)) * 4 * *(anchors + 2 * anchor_idx + 0) / input_w;  // [0, 1]
        det->bbox[3] = square(*(cur_input + 3 * total_grids)) * 4 * *(anchors + 2 * anchor_idx + 1) / input_h;  // [0, 1]

        det->bbox[0] -= det->bbox[2] / 2;  // shift from center to top-left
        det->bbox[1] -= det->bbox[3] / 2;

        det->det_confidence = box_prob;
        det->class_id = class_id;
        det->class_confidence = max_cls_prob;
    }

    void YoloPluginDynamic::forwardGpu(const float* const* inputs, float* output, hipStream_t stream, int batchSize)
    {
        int num_elements = batchSize * mNumAnchors * mYoloWidth * mYoloHeight;

        //CHECK(hipMemset(output, 0, num_elements * sizeof(Detection)));

        if (mNewCoords) {
            CalDetection_NewCoords<<<(num_elements + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream>>>
                (inputs[0], output, batchSize, mYoloWidth, mYoloHeight, mNumAnchors, (const float*) mAnchors, mNumClasses, mInputWidth, mInputHeight, mScaleXY);
        } else {
            CalDetection<<<(num_elements + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream>>>
                (inputs[0], output, batchSize, mYoloWidth, mYoloHeight, mNumAnchors, (const float*) mAnchors, mNumClasses, mInputWidth, mInputHeight, mScaleXY);
        }
    }

    int YoloPluginDynamic::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream)
    {
        const int batchSize = inputDesc->dims.d[0];
        forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    YoloPluginDynamicCreator::YoloPluginDynamicCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloPluginDynamicCreator::getPluginName() const
    {
        return "YoloPluginDynamic_TRT";
    }

    const char* YoloPluginDynamicCreator::getPluginVersion() const
    {
        return "1";
    }

    const PluginFieldCollection* YoloPluginDynamicCreator::getFieldNames()
    {
        return &mFC;
    }

    IPluginV2* YoloPluginDynamicCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        ASSERT(!strcmp(name, getPluginName()));
        const PluginField* fields = fc->fields;
        int yolo_width, yolo_height, num_anchors = 0;
        float anchors[MAX_ANCHORS * 2];
        int num_classes, input_multiplier, new_coords = 0;
        float scale_x_y = 1.0;

        for (int i = 0; i < fc->nbFields; ++i)
        {
            const char* attrName = fields[i].name;
            if (!strcmp(attrName, "yoloWidth"))
            {
                ASSERT(fields[i].type == PluginFieldType::kINT32);
                yolo_width = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "yoloHeight"))
            {
                ASSERT(fields[i].type == PluginFieldType::kINT32);
                yolo_height = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "numAnchors"))
            {
                ASSERT(fields[i].type == PluginFieldType::kINT32);
                num_anchors = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "numClasses"))
            {
                ASSERT(fields[i].type == PluginFieldType::kINT32);
                num_classes = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "inputMultiplier"))
            {
                ASSERT(fields[i].type == PluginFieldType::kINT32);
                input_multiplier = *(static_cast<const int*>(fields[i].data));
            }
            else if (!strcmp(attrName, "anchors")){
                ASSERT(num_anchors > 0 && num_anchors <= MAX_ANCHORS);
                ASSERT(fields[i].type == PluginFieldType::kFLOAT32);
                memcpy(anchors, static_cast<const float*>(fields[i].data), num_anchors * 2 * sizeof(float));
            }
            else if (!strcmp(attrName, "scaleXY"))
            {
                ASSERT(fields[i].type == PluginFieldType::kFLOAT32);
                scale_x_y = *(static_cast<const float*>(fields[i].data));
            }
            else if (!strcmp(attrName, "newCoords"))
            {
                ASSERT(fields[i].type == PluginFieldType::kINT32);
                new_coords = *(static_cast<const int*>(fields[i].data));
            }
            else
            {
                std::cerr <<  "Unknown attribute: " << attrName << std::endl;
                ASSERT(0);
            }
        }
        ASSERT(yolo_width > 0 && yolo_height > 0);
        ASSERT(anchors[0] > 0.0f && anchors[1] > 0.0f);
        ASSERT(num_classes > 0);
        ASSERT(input_multiplier == 8 || input_multiplier == 16 || input_multiplier == 32);
        ASSERT(scale_x_y >= 1.0);

        YoloPluginDynamic* obj = new YoloPluginDynamic(name, yolo_width, yolo_height, num_anchors, anchors, num_classes, yolo_width * input_multiplier, yolo_height * input_multiplier, scale_x_y, new_coords);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2* YoloPluginDynamicCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        YoloPluginDynamic* obj = new YoloPluginDynamic(name, serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    PluginFieldCollection YoloPluginDynamicCreator::mFC{};
    std::vector<PluginField> YoloPluginDynamicCreator::mPluginAttributes;
    REGISTER_TENSORRT_PLUGIN(YoloPluginDynamicCreator);
} // namespace nvinfer1
